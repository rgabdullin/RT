#include "hip/hip_runtime.h"
#include "Sampler.h"

/* Sampler */
__device__
void Sampler::init(int num_of_samples, int num_of_sets, float e) {
	GenerateSamples(num_of_samples, num_of_sets);

	MapSamplesToHemisphere(e);
}

__device__
void Sampler::MapSamplesToHemisphere(float e) {
	hemisphere_samples = (float3*)malloc(sizeof(float3) * num_samples * num_sets);

	for (int i = 0; i < num_samples * num_sets; ++i) {
		float3 tmp;
		float sin_phi = sin(2.0f * acos(-1.0f) * square_samples[i].x);
		float cos_phi = cos(2.0f * acos(-1.0f) * square_samples[i].x);
		float cos_theta = pow((1.0f - square_samples[i].y), 1.0f / (e + 1.0f));
		float sin_theta = sqrt(1.0f - cos_theta * cos_theta);
		tmp = make_float3(sin_theta * cos_phi, sin_theta * sin_phi, cos_theta);
		hemisphere_samples[i] = tmp;
	}
}

__device__
float2 Sampler::SampleUnitSquare(int l_num_of_sample, int l_num_of_set) {
	return square_samples[num_samples * (l_num_of_set % num_sets) + (l_num_of_sample % num_samples)];
}

__device__
float3 Sampler::SampleHemisphere(int l_num_of_sample, int l_num_of_set) {
	return hemisphere_samples[num_samples * (l_num_of_set % num_sets) + (l_num_of_sample % num_samples)];
}

__device__
Sampler::~Sampler(void) {
	free(square_samples);
	free(hemisphere_samples);
}
