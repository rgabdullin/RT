#include "hip/hip_runtime.h"
#include "MultiJitteredSampler.h"
#include "hiprand.h"
#include "hiprand/hiprand_kernel.h"

/* MultiJitteredSampler */
__device__
void MultiJitteredSampler::shuffle_coordinates(int offset) {
	hiprandState_t curand_state;
	hiprand_init(23041996,0,0,&curand_state);

	for (int i = 0; i < num_samples; ++i) {
		int k = hiprand(&curand_state) % num_samples;
		int j = hiprand(&curand_state) % num_samples;
		float c;
		c = square_samples[i + offset].x;
		square_samples[i + offset].x = square_samples[k + offset].x;
		square_samples[k + offset].x = c;
		c = square_samples[i + offset].y;
		square_samples[i + offset].y = square_samples[j + offset].y;
		square_samples[j + offset].y = c;
	}
}
__device__
void MultiJitteredSampler::GenerateSamples(int l_num_of_samples, int l_num_of_sets) {
	hiprandState_t curand_state;
	hiprand_init(23041996,0,0,&curand_state);

	square_samples = (float2*)malloc(sizeof(float2) * l_num_of_samples * l_num_of_sets);

	int n = sqrt(float(l_num_of_samples));
	num_samples = n*n;
	num_sets = l_num_of_sets;

	for (int i = 0; i < l_num_of_sets; ++i) {
		for (int y = 0; y < n; ++y)
			for (int x = 0; x < n; ++x)
				square_samples[i * num_samples + n * y + x] = make_float2(
					float(x) / n + (float(y) / n + hiprand_uniform(&curand_state) / n) / n,
					float(y) / n + (float(x) / n + hiprand_uniform(&curand_state) / n) / n);
		shuffle_coordinates(i * num_samples);
	}
}
__device__
MultiJitteredSampler::~MultiJitteredSampler() {}
