#include "hip/hip_runtime.h"
#include "JitteredSampler.h"
#include "hiprand.h"
#include "hiprand/hiprand_kernel.h"

/* JitteredSampler */
__device__
void JitteredSampler::GenerateSamples(int l_num_of_samples, int l_num_of_sets) {
	hiprandState_t curand_state;
	hiprand_init(23041996,0,0,&curand_state);

	square_samples = (float2*)malloc(sizeof(float2) * l_num_of_samples * l_num_of_sets);

	int n = sqrt(float(l_num_of_samples));
	num_samples = n*n;
	num_sets = l_num_of_sets;

	for (int i = 0; i < l_num_of_sets; ++i)
		for (int y = 0; y < n; ++y)
			for (int x = 0; x < n; ++x)
				square_samples[i * num_samples + n * y + x] = make_float2(float(x) / n + hiprand_uniform(&curand_state) / n, float(y) / n + hiprand_uniform(&curand_state) / n);
}
__device__
JitteredSampler::~JitteredSampler() {}
