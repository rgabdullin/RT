#include "hip/hip_runtime.h"
#include "RegularSampler.h"

/* RegularSampler */
__device__
void RegularSampler::GenerateSamples(int l_num_of_samples, int l_num_of_sets) {
	square_samples = (float2*)malloc(sizeof(float2) * l_num_of_samples * l_num_of_sets);
	int n = sqrt(float(l_num_of_samples));
	num_samples = n*n;
	num_sets = l_num_of_sets;
	for (int i = 0; i < l_num_of_sets; ++i)
		for (int y = 0; y < n; ++y)
			for (int x = 0; x < n; ++x)
				square_samples[i * num_samples + n * y + x] = make_float2(float(x) / n + 1.0 / (2 * n), float(y) / n + 1.0 / (2 * n));
}
__device__
RegularSampler::~RegularSampler() {}
