#include "hip/hip_runtime.h"
#include "../Utils/HitRec.h"
#include "../Utils/World.h"
#include "Phong.h"
#include "Material.h"

/* Phong */
__device__
Phong::Phong():
	ambient_brdf(new Lambertian()),
	diffuse_brdf(new Lambertian()),
	specular_brdf(new GlossySpecular())
{}
__device__
Phong::Phong(Sampler* sampler_ptr, float ka, float kd, float ks, float exp, float3 color) :
	ambient_brdf(new Lambertian()),
	diffuse_brdf(new Lambertian()),
	specular_brdf(new GlossySpecular())
{
	init(sampler_ptr, ka, kd, ks, exp, color);
}
__device__
void Phong::init(Sampler* sampler_ptr, float ka, float kd, float ks, float exp, float3 color){
	ambient_brdf->SetSampler(sampler_ptr);
	ambient_brdf->Set_kd(ka);
	ambient_brdf->Set_color(color);

	diffuse_brdf->SetSampler(sampler_ptr);
	diffuse_brdf->Set_kd(kd);
	diffuse_brdf->Set_color(color);

	specular_brdf->SetSampler(sampler_ptr);
	specular_brdf->Set_ks(ks);
	specular_brdf->Set_exp(exp);
	specular_brdf->Set_color(color);
}
__device__
float3 Phong::shade(HitRec* hr) {
	float3 wo = -1.0f * hr->ray.direction;
	float3 L = ambient_brdf->rho(hr, wo) * hr->wr->ambient_ptr->L(hr);

	int num_of_lights = hr->wr->num_of_lights;

	Ray ray = hr->ray;
	HitRec sr;
	for (int i = 0; i < num_of_lights; ++i) {
		make_HitRec(&sr, hr->wr, &ray);
		sr.ray.direction = normalize(hr->wr->scene_lights[i]->GetDirection(hr));
		sr.ray.origin = hr->hit_point;
		hr->wr->isHitSceneObject(&sr.ray, &sr);
		if (sr.isHit == false) {
			float3 wi = sr.ray.direction;
			float ndotwi = dot(hr->hit_normal, wi);
			if (ndotwi < 0)
				ndotwi = dot(-hr->hit_normal, wi);
			L += (diffuse_brdf->f(hr, wo, wi) + specular_brdf->f(hr, wo, wi)) * hr->wr->scene_lights[i]->L(hr) * ndotwi;
		}
	}
	return L;
}

__device__
Phong::~Phong() {
	if (ambient_brdf != NULL)
		delete ambient_brdf;
	if (diffuse_brdf != NULL)
		delete diffuse_brdf;
	if (specular_brdf != NULL)
		delete specular_brdf;
}
