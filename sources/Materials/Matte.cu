#include "hip/hip_runtime.h"
#include "../Utils/HitRec.h"
#include "../Utils/World.h"
#include "Matte.h"
#include "Material.h"

/* Matte */
__device__
Matte::Matte():
	ambient_brdf(new Lambertian()),
	diffuse_brdf(new Lambertian())
{}
__device__
Matte::Matte(Sampler* sampler_ptr, float ka, float kd, float3 color) :
	ambient_brdf(new Lambertian()),
	diffuse_brdf(new Lambertian())
{
	init(sampler_ptr, ka, kd, color);
}
__device__
void Matte::init(Sampler* sampler_ptr, float ka, float kd, float3 color){
	ambient_brdf->SetSampler(sampler_ptr);
	ambient_brdf->Set_kd(ka);
	ambient_brdf->Set_color(color);

	diffuse_brdf->SetSampler(sampler_ptr);
	diffuse_brdf->Set_kd(kd);
	diffuse_brdf->Set_color(color);
}
__device__
float3 Matte::shade(HitRec* hr) {
	float3 wo = -1.0f * hr->ray.direction;
	float3 L = ambient_brdf->rho(hr, wo) * hr->wr->ambient_ptr->L(hr);

	int num_of_lights = hr->wr->num_of_lights;

	Ray ray = hr->ray;
	HitRec sr;
	for (int i = 0; i < num_of_lights; ++i) {
		make_HitRec(&sr, hr->wr, &ray);
		sr.ray.direction = hr->wr->scene_lights[i]->GetDirection(hr);
		sr.ray.origin = hr->hit_point;
		hr->wr->isHitSceneObject(&sr.ray, &sr);
		if (sr.isHit == false) {
			float3 wi = sr.ray.direction;
			float ndotwi = dot(hr->hit_normal, wi);
			if (ndotwi < 0)
				ndotwi = dot(-hr->hit_normal, wi);
			L += diffuse_brdf->f(hr, wo, wi) * hr->wr->scene_lights[i]->L(hr) * ndotwi;
		}
	}
	return L;
}

__device__
Matte::~Matte() {
	if (ambient_brdf != NULL)
		delete ambient_brdf;
	if (diffuse_brdf != NULL)
		delete diffuse_brdf;
}
