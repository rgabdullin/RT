#include "hip/hip_runtime.h"
#include "Lambertian.h"
#include "../Samplers.h"
#include "../Utils/HitRec.h"

/* Lambertian */

__device__
void Lambertian::SetSampler(Sampler* ptr) {
	sampler_ptr = ptr;
}
__device__
void Lambertian::Set_kd(float l_kd){
	kd = l_kd;
}
__device__
void Lambertian::Set_color(float3 l_color) {
	color = l_color;
}

__device__
float3 Lambertian::f(HitRec* hr, float3& wi, float3& wo) {
	return kd * color / M_PI;
}

__device__
float3 Lambertian::sample_f(HitRec * hr, float3 & wi, float3 & wo) {
	return make_float3(0.0f,0.0f,0.0f);
}

__device__
float3 Lambertian::rho(HitRec* hr, float3& wo) {
	return kd * color;
}

__device__
Lambertian::~Lambertian() {}
