#include "hip/hip_runtime.h"
#include "GlossySpecular.h"
#include "../Samplers.h"
#include "../Utils/HitRec.h"
#include "../Utils/cuda_double_math.h"

/* GlossySpecular */

__device__
void GlossySpecular::SetSampler(Sampler* ptr) {
	sampler_ptr = ptr;
}
__device__
void GlossySpecular::Set_ks(float l_ks){
	ks = l_ks;
}
__device__
void GlossySpecular::Set_exp(float l_exp){
	exp = l_exp;
}
__device__
void GlossySpecular::Set_color(float3 l_color) {
	color = l_color;
}

__device__
float3 GlossySpecular::f(HitRec* hr, float3& wi, float3& wo) {
	float3 col = make_float3(0.0f,0.0f,0.0f);
	float ndotwi = dot(hr->hit_normal, wi);
	float3 r = -1.0f * wi + 2.0f * hr->hit_normal * ndotwi;
	float rdotwo = dot(r, wo);

	if(rdotwo > 0.0f)
		col = color * pow(rdotwo, exp);
	return col;
}

__device__
float3 GlossySpecular::sample_f(HitRec * hr, float3 & wi, float3 & wo) {
	return make_float3(0.0f,0.0f,0.0f);
}

__device__
float3 GlossySpecular::rho(HitRec* hr, float3& wo) {
	return make_float3(0.0f,0.0f,0.0f);
}

__device__
GlossySpecular::~GlossySpecular() {}


