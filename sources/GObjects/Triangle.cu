#include "hip/hip_runtime.h"
#include "Triangle.h"
#include "../Utils/cuda_math.h"
#include "../Utils/cuda_double_math.h"
#include "../Materials.h"

/* Triangle */

__device__
Triangle::Triangle() {}

__device__
void Triangle::init(float3 p1, float3 p2, float3 p3, Material* material_ptr) {
	points[0] = p1;
	points[1] = p2;
	points[2] = p3;
	normal = normalize(cross(p1 - p2, p3 - p2));
	SetMaterial(material_ptr);
}

__device__
bool Triangle::Intersection(const Ray& ray, HitRec& hr) {
	__shared__ double3 p0; if (threadIdx.x == 0) p0 = make_double3(points[0]);
	__shared__ double3 p1; if (threadIdx.x == 0) p1 = make_double3(points[1]);
	__shared__ double3 p2; if (threadIdx.x == 0) p2 = make_double3(points[2]);
	__shared__ double3 n; if (threadIdx.x == 0)  n = make_double3(normal);
	__syncthreads();
	double3 ro = make_double3(ray.origin);
	double3 dir = make_double3(ray.direction);
	double t = dot(p1 - ro, n) / dot(dir, n);
	double3 hp = ro + t * dir;

	if (t > eps && (dot(cross(p1 - p0, hp - p0), cross(p2 - p1, hp - p1)) >= 0
		&& dot(cross(p2 - p1, hp - p1), cross(p0 - p2, hp - p2)) >= 0)
		&& (t < hr.tmin || !hr.isHit)) {
		hr.isHit = true;
		hr.tmin = t;

		hr.hit_point = make_float3(hp);
		hr.hit_normal = normalize(normal);
		hr.ray = ray;
		hr.material_ptr = material_ptr;

		return true;
	}
	return false;
}

__device__
Triangle::~Triangle() {}
