#include "hip/hip_runtime.h"
#include "Plane.h"
#include "../Utils/cuda_math.h"
#include "../Utils/cuda_double_math.h"
#include "../Materials.h"

/* Plane */

__device__
Plane::Plane() {}

__device__
void Plane::init(float3 p, float3 n, Material* material_ptr) {
	point = p;
	normal = normalize(n);
	SetMaterial(material_ptr);
}

__device__
bool Plane::Intersection(const Ray& ray, HitRec& hr) {
	__shared__ double3 p; if (threadIdx.x == 0) p = make_double3(point);
	__shared__ double3 n; if (threadIdx.x == 0) n = make_double3(normal);
	__syncthreads();
	double3 ro = make_double3(ray.origin);
	double3 dir = make_double3(ray.direction);
	double t = dot(p - ro, n) / dot(dir, n);
	if (t > eps && (t < hr.tmin || !hr.isHit)) {
		hr.isHit = true;
		hr.tmin = t;

		hr.hit_point = ray.origin + t * ray.direction;
		hr.hit_normal = normal;
		hr.ray = ray;
		hr.material_ptr = material_ptr;

		return true;
	}
	return false;
}
__device__
Plane::~Plane() {}
