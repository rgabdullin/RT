#include "hip/hip_runtime.h"
#include "Chessplate.h"
#include "../Utils/cuda_math.h"
#include "../Utils/cuda_double_math.h"
#include "../Materials.h"

/* Chessplate */

__device__
Chessplate::Chessplate() {}

__device__
void Chessplate::init(float3 p, float3 u, float3 r, float2 l_size, float l_cell_size, Material* material_ptr) {
	center = p;
	up = normalize(u);
	right = normalize(r);
	size = l_size;
	cell_size = l_cell_size;
	SetMaterial(material_ptr);
}

__device__
bool Chessplate::Intersection(const Ray& ray, HitRec& hr) {
	__shared__ double3 p; if (threadIdx.x == 0) p = make_double3(center);
	__shared__ double3 n; if (threadIdx.x == 0) n = make_double3(cross(right, up));
	__syncthreads();
	double3 ro = make_double3(ray.origin);
	double3 dir = make_double3(ray.direction);
	double t = dot(p - ro, n) / dot(dir, n);
	if (t > eps) {
		double3 vec = ro + t * dir - (make_double3(center) - size.y * 0.5 * make_double3(up) - size.x * 0.5 * make_double3(right));
		double pr_up = dot(make_double3(up), vec);
		double pr_right = dot(make_double3(right), vec);
		int2 cell = make_int2((int)(pr_right / cell_size),(int)(pr_up / cell_size));
		if(pr_up >= 0.0f && pr_right >= 0.0f && pr_up <= size.y && pr_right <= size.x && (cell.x + cell.y) % 2 == 0) {
			if(t < hr.tmin || !hr.isHit) {
				hr.isHit = true;
				hr.tmin = t;

				hr.hit_point = ray.origin + t * ray.direction;
				hr.hit_normal = normalize(cross(right, up));
				hr.ray = ray;
				hr.material_ptr = material_ptr;

				return true;
			}
		}
	}
	return false;
}
__device__
Chessplate::~Chessplate() {}
