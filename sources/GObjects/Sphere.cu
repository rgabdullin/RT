#include "hip/hip_runtime.h"
#include "Sphere.h"
#include "../Utils/cuda_math.h"
#include "../Utils/cuda_double_math.h"
#include "../Materials.h"

/* Sphere */
__device__
Sphere::Sphere() :
	origin(make_float3(0, 0, 0)),
	radius(1)
{}

__device__
void Sphere::init(float3 l_origin, float l_radius, Material* material_ptr) {
	origin = l_origin;
	radius = l_radius;
	SetMaterial(material_ptr);
}

__device__
bool Sphere::Intersection(const Ray& ray, HitRec& hr) {
	double3 dir = make_double3(ray.direction);
	double3 ro = make_double3(ray.origin);
	double3 o = make_double3(origin);
	double r = radius;
	double A = dot(dir, dir);
	double B = 2.0f * dot(ro - o, dir);
	double C = dot(ro - o, ro - o) - r * r;

	double D = B * B - 4 * A * C;
	if (D >= 0) {
		double t1 = (-B - sqrt(D)) / A * 0.5;
		double t2 = (-B + sqrt(D)) / A * 0.5;
		double t = -1;
		if (t1 > eps)
			t = t1;
		else
			if (t2 > eps)
				t = t2;
		if (t > eps && (!hr.isHit || t < hr.tmin)) {
			hr.isHit = true;
			hr.tmin = t;

			hr.hit_point = ray.origin + ray.direction * t;
			hr.hit_normal = normalize(hr.hit_point - origin);
			hr.ray = ray;
			hr.material_ptr = material_ptr;

			return true;
		}
	}
	return false;
}

__device__
Sphere::~Sphere() {}
