#include "hip/hip_runtime.h"
#include "Plate.h"
#include "../Utils/cuda_math.h"
#include "../Utils/cuda_double_math.h"
#include "../Materials.h"

/* Plate */

__device__
Plate::Plate() {}

__device__
void Plate::init(float3 p, float3 u, float3 r, float2 l_size, Material* material_ptr) {
	center = p;
	up = normalize(u);
	right = normalize(r);
	size = l_size;
	SetMaterial(material_ptr);
}

__device__
bool Plate::Intersection(const Ray& ray, HitRec& hr) {
	__shared__ double3 p; if (threadIdx.x == 0) p = make_double3(center);
	__shared__ double3 n; if (threadIdx.x == 0) n = make_double3(cross(right, up));
	__syncthreads();
	double3 ro = make_double3(ray.origin);
	double3 dir = make_double3(ray.direction);
	double t = dot(p - ro, n) / dot(dir, n);
	if (t > eps) {
		double3 vec = ro + t * dir - (make_double3(center) - size.y * 0.5 * make_double3(up) - size.x * 0.5 * make_double3(right));
		double pr_up = dot(make_double3(up), vec);
		double pr_right = dot(make_double3(right), vec);
		if(pr_up >= 0.0f && pr_right >= 0.0f && pr_up <= size.y && pr_right <= size.x) {
			if(t < hr.tmin || !hr.isHit) {
				hr.isHit = true;
				hr.tmin = t;

				hr.hit_point = ray.origin + t * ray.direction;
				hr.hit_normal = normalize(cross(right, up));
				hr.ray = ray;
				hr.material_ptr = material_ptr;

				return true;
			}
		}
	}
	return false;
}
__device__
Plate::~Plate() {}
