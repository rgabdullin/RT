#include "hip/hip_runtime.h"
#ifndef SYNC_AND_CHECK_CUDA_ERRORS
#define SYNC_AND_CHECK_CUDA_ERRORS {hipStreamSynchronize(0); hipError_t x = hipGetLastError(); if ((x) != hipSuccess) { printf("Error: %s\n", hipGetErrorString(x)); fclose(stdout); exit(1); }}
#endif

#include <cstdio>
#include <cstdlib>
#include <sstream>
#include <ctime>
#include <string>
#include <hip/hip_runtime.h>
#include <>
#include "hiprand.h"
#include "hiprand/hiprand_kernel.h"

#include "./Utils/World.h"


void printInfo(void) {
	time_t start_time; time(&start_time);
	struct tm* s_time = localtime(&start_time);
	printf("Runnning. time = %04d/%02d/%02d %02d:%02d:%02d\n",
			s_time->tm_year + 1900,
			s_time->tm_mon + 1,
			s_time->tm_mday,
			s_time->tm_hour,
			s_time->tm_min,
			s_time->tm_sec);

	hipDeviceProp_t props;
	int dev_count;

	hipGetDeviceCount(&dev_count);

	printf("Detected %d devices:\n", dev_count);
	for (int i = 0; i < dev_count; ++i) {
		hipGetDeviceProperties(&props, i);
		printf("\t[ %d ] %s, %.1f GBs memory, CUDA %d.%d Compute Capability\n", i, props.name, float(props.totalGlobalMem) / (1024 * 1024 * 1024), props.major, props.minor);
	}
	printf("\n");
	SYNC_AND_CHECK_CUDA_ERRORS;
}

int main(int argc, char** argv) {
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	clock_t start_clock, end_clock;

	printInfo();

	int framesize = 256 * 256;
	if (argc > 1)
		sscanf(argv[1], "%d", &framesize);
	start_clock = clock();

	World* w;
	hipMallocManaged(&w, sizeof(World));

	initWorld(w, 4 * make_int2(512, 256), 4, 16);
	buildScene(w);

	//Tracing
	hipEventRecord(start, 0);

	renderScene(w, framesize);

	hipEventRecord(stop, 0);
	hipStreamSynchronize(0);

	float _time;
	hipEventElapsedTime(&_time, start, stop);
	printf("Tracing time: %.2f ms\n\n", _time);

	//Saving image
	std::stringstream st;
	time_t now_oclock; time(&now_oclock);
	clock_t now_clock = clock();
	st << "./pics/pic" << now_oclock << "_" << now_clock % 1000 << ".bmp";
	char filename[255];
	st.getline(filename, 255);
	//saveSceneToFile(w, filename);
	saveSceneToFile(w, "./pics/test.bmp");
	clearScene(w);

	end_clock = clock();
	printf("TIME ELAPSED: %lf\n", (end_clock - start_clock) / 1000.0);

	fclose(stdout);

	return 0;
}
