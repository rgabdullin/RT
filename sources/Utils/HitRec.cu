#include "hip/hip_runtime.h"
#include "HitRec.h"
#include "World.h"
#include "../Materials.h"
#include "Ray.h"

/* HitRec */
__host__ __device__
void make_HitRec(HitRec* hr, World* wr, Ray* ray) {
	hr->isHit = false;

	hr->hit_point = make_float3(0.0f, 0.0f, 0.0f);
	hr->hit_normal = make_float3(0.0f, 0.0f, 0.0f);
	hr->ray = *ray;
	hr->wr = wr;
	hr->material_ptr = NULL;

	hr->tmin = 1e8f;
}
