#include "hip/hip_runtime.h"
#include "ViewPlane.h"

/* ViewPlane */
__host__ __device__
ViewPlane::ViewPlane(int2 res, float height){
	init(res, height);
}
__host__ __device__
void ViewPlane::init(int2 l_res, float l_height) {
	res = l_res;
	psize = l_height / (res.y + 1);
}
