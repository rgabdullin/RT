#include "hip/hip_runtime.h"
#include "World.h"
#include "../Utils/FreeImage.h"
#include "../Utils/FreeImagePlus.h"
#include <cstdio>

void saveSceneToFile(World *w, std::string filename) {
	printf("Saving image in file \"%s\"\n\t", filename.c_str()); fflush(stdout);
	fipImage img(FIT_BITMAP, w->vp.res.x, w->vp.res.y, 24);
	BYTE* ptr = img.accessPixels();
	int pitch = img.getScanWidth();
	int bmask = FreeImage_GetBlueMask(img);
	int rmask = FreeImage_GetRedMask(img);
	int r = 0, b = 2;
	if (rmask > bmask) {
		r = 2; b = 0;
	}
	float3 a0 = make_float3(0, 0, 0);
	float3 a1 = make_float3(1, 1, 1);
	for(int k = 0; k < w->vp.res.y; ++k)
		for (int i = 0; i < w->vp.res.x; ++i) {
			float3 v = clamp(w->image[i + k * w->vp.res.x], a0, a1) * 255.0f;
			ptr[pitch * k + 3 * i + r] = (unsigned char)(v.x);
			ptr[pitch * k + 3 * i + 1] = (unsigned char)(v.y);
			ptr[pitch * k + 3 * i + b] = (unsigned char)(v.z);
		}
	img.save(filename.c_str());
	img.clear();
	printf("OK\n"); fflush(stdout);
}
