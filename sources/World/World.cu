#include "hip/hip_runtime.h"
#include "World.h"

#include <cstdio>
#include "hiprand.h"
#include "hiprand/hiprand_kernel.h"
#include "FreeImagePlus.h"

/* World */
__device__
World::World() :
	image(NULL),
	gpu_buffer(NULL),
	num_rays_per_pixel(1),
	background_color(make_float3(0.0f,0.0f,0.25f)),
	num_of_objects(1),
	scene_objs(NULL),
	num_of_lights(1),
	scene_lights(NULL),
	ambient_ptr(NULL),
	num_of_materials(1),
	scene_materials(NULL),
	camera(NULL),
	ray_tracer(NULL),
	pixel_sampler(NULL)
{
	vp.init(make_int2(1024,512), 4);
}

__device__
bool World::isHitSceneObject(Ray * ray, HitRec * hr) {
	for (int i = 0; i < num_of_objects; ++i)
		scene_objs[i]->Intersection(*ray, *hr);

	return hr->isHit;
}
