#include "hip/hip_runtime.h"
#include "World.h"
#include <cstdio>

__global__
void ClearMaterials(Material **ptr) {
	int idx = threadIdx.x;

	delete ptr[idx];
}
__global__
void ClearObjects(GObject **ptr) {
	int idx = threadIdx.x;

	delete ptr[idx];
}
__global__
void ClearLights(Light **ptr) {
	int idx = threadIdx.x;

	delete ptr[idx];
}
__global__
void ClearGPU(World *ptr) {
	delete ptr->camera;
	delete ptr->ray_tracer;
	delete ptr->pixel_sampler;
	delete ptr->ambient_ptr;
}

void clearScene(World *wr) {
	printf("Destroying scene\n"); fflush(stdout);
	free(wr->image);

	if(wr->num_of_objects > 0) {
		printf("\t-Objects\n"); fflush(stdout);
		ClearObjects <<< 1, wr->num_of_objects >>> (wr->scene_objs);
	}
	SYNC_AND_CHECK_CUDA_ERRORS;

	if (wr->num_of_materials > 0) {
		printf("\t-Materials\n"); fflush(stdout);
		ClearMaterials <<< 1, wr->num_of_materials >>> (wr->scene_materials);
	}
	SYNC_AND_CHECK_CUDA_ERRORS;

	if (wr->num_of_lights > 0) {
		printf("\t-Lights\n"); fflush(stdout);
		ClearLights <<< 1, wr->num_of_lights >>> (wr->scene_lights);
	}
	SYNC_AND_CHECK_CUDA_ERRORS;

	ClearGPU <<< 1, 1 >>> (wr);
	SYNC_AND_CHECK_CUDA_ERRORS;

	hipFree(wr->scene_materials);
	hipFree(wr->scene_lights);
	hipFree(wr->scene_objs);

	SYNC_AND_CHECK_CUDA_ERRORS;

	hipFree(wr);

	SYNC_AND_CHECK_CUDA_ERRORS;
	printf("\tOK\n"); fflush(stdout);
}
