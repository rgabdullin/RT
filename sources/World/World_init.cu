#include "hip/hip_runtime.h"
#include "World.h"
#include <cstdio>

__global__
void initWorld_gpu(World * w) {
	w->pixel_sampler = new MultiJitteredSampler();
	w->pixel_sampler->init(w->num_rays_per_pixel, 67);

	w->ray_tracer = new PrimaryRayTracer(w);
}

void initWorld(World *w, int2 res, float size, int num_rays_per_pixel) {
	w->vp.init(res, size);

	w->num_rays_per_pixel = num_rays_per_pixel;

	w->background_color = make_float3(0, 0, 0.25);

	w->image = (float3*)malloc(sizeof(float3) * res.x * res.y);
	printf("Allocated memory to image: %.2fKBs\n", float(sizeof(float3) * res.x * res.y) / 1024); fflush(stdout);
	SYNC_AND_CHECK_CUDA_ERRORS;

	initWorld_gpu <<< 1, 1 >>>(w);
	SYNC_AND_CHECK_CUDA_ERRORS;
}
