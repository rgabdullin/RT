#include "hip/hip_runtime.h"
#include "World.h"
#include <cstdio>

__global__
void buildScene1_gpu(World * w) {
	w->num_of_objects = 2;
	w->num_of_lights = 3;
	w->num_of_materials = w->num_of_objects;

	w->scene_objs = (GObject**)malloc(sizeof(GObject*) * w->num_of_objects);
	w->scene_lights = (Light**)malloc(sizeof(Light*) * w->num_of_lights);
	w->scene_materials = (Material**)malloc(sizeof(Material*) * w->num_of_materials);

	PinholeCamera *c = new PinholeCamera(w);
	c->init(make_float3(0, 0, 25), make_float3(0, 0, 0), make_float3(0, 1, 0));
	c->init_pinhole(4,1);
	w->camera = c;

	w->ambient_ptr = new AmbientLight(0.3f);

	PointLight* lt1 = new PointLight(3.0f * make_float3(-5.0f, 12.0f, 4.0f), 0.8f);
	w->scene_lights[0] = lt1;

	PointLight* lt2 = new PointLight(2.0f * make_float3(4.0f, 10.0f, -1.5f), 1.0f);
	w->scene_lights[1] = lt2;

	DirectionalLight* lt3 = new DirectionalLight(make_float3(-1.0f, -1.0f, -2.0f), 0.7f);
	w->scene_lights[2] = lt3;

	Matte* mt1 = new Matte(w->pixel_sampler, 0.3f, 0.7f, make_float3(0.6f, 0.6f, 0.6f));
	w->scene_materials[0] = mt1;
	Plane * ptr1 = new Plane();
	ptr1->init(make_float3(0, -8.0f, 0), make_float3(0, 1, 0), mt1);
	w->scene_objs[0] = ptr1;

	Phong* mt2 = new Phong(w->pixel_sampler, 0.3f, 0.8f, 0.1f, 15.0f, make_float3(0.9f, 0.001f, 0.001f));
	w->scene_materials[1] = mt2;
	Sphere * ptr2 = new Sphere();
	ptr2->init(make_float3(0, 0, 0), 4, mt2);
	w->scene_objs[1] = ptr2;
}

__global__
void buildScene2_gpu(World * w) {
	w->num_of_objects = 3;
	w->num_of_lights = 3;
	w->num_of_materials = w->num_of_objects;

	w->scene_objs = (GObject**)malloc(sizeof(GObject*) * w->num_of_objects);
	w->scene_lights = (Light**)malloc(sizeof(Light*) * w->num_of_lights);
	w->scene_materials = (Material**)malloc(sizeof(Material*) * w->num_of_materials);

	PinholeCamera *c = new PinholeCamera(w);
	c->init(make_float3(-16, 0, 25), make_float3(0, 0, 0), make_float3(0, 1, 0));
	c->init_pinhole(4,1);
	w->camera = c;

	w->ambient_ptr = new AmbientLight(0.3f);

	PointLight* lt1 = new PointLight(make_float3(-0.5,0,25), 0.4f);
	w->scene_lights[0] = lt1;

	PointLight* lt2 = new PointLight(make_float3( 0.7,0,25), 0.4f);
	w->scene_lights[1] = lt2;

	DirectionalLight* lt3 = new DirectionalLight(make_float3(-8, 0, -25), 1.0f);
	w->scene_lights[2] = lt3;

	Matte* mt1 = new Matte(w->pixel_sampler, 0.3f, 0.7f, make_float3(1.0f, 1.0f, 1.0f));
	w->scene_materials[0] = mt1;
	Plate * ptr1 = new Plate();
	ptr1->init(make_float3(0, 0, 0), make_float3(0, 1, 0), make_float3(1, 0, 0), make_float2(33,25), mt1);
	w->scene_objs[0] = ptr1;

	Matte* mt2 = new Matte(w->pixel_sampler, 0.3f, 0.8f, make_float3(0.01f, 1.0f, 0.01f));
	w->scene_materials[1] = mt2;
	Chessplate * ptr2 = new Chessplate();
	ptr2->init(make_float3(0, 0, 19.9), make_float3(0, 1, 0), make_float3(1, 0, 0.0), make_float2(12.5,8.5), 0.1f, mt2);
	w->scene_objs[1] = ptr2;

	Matte* mt3 = new Matte(w->pixel_sampler, 0.3f, 0.8f, make_float3(1.0f, 0.01f, 0.01f));
	w->scene_materials[2] = mt3;
	Chessplate* ptr3 = new Chessplate();
	ptr3->init(make_float3(0, 0, 20.0), make_float3(0, 1, 0), make_float3(1, 0, 0.0), make_float2(12.5,8.5), 0.5f / 7, mt3);
	w->scene_objs[2] = ptr3;
}

void buildScene(World *w) {
	printf("Building scene\n"); fflush(stdout);

	buildScene2_gpu <<< 1, 1 >>> (w);
	SYNC_AND_CHECK_CUDA_ERRORS;

	printf("\tOK\n"); fflush(stdout);
}
