#include "hip/hip_runtime.h"
#include "World.h"
#include <cstdio>

__global__
void buildScene_gpu(World * w) {
	PinholeCamera *c = new PinholeCamera(w);
	c->init(make_float3(0, 0, 25), make_float3(0, 0, 0), make_float3(0, 1, 0));
	c->init_pinhole(4,1);
	w->camera = c;

	w->ambient_ptr = new AmbientLight(0.3f);

	PointLight* lt1 = new PointLight(3.0f * make_float3(-5.0f, 12.0f, 4.0f), 1.5f);
	w->scene_lights[0] = lt1;

	PointLight* lt2 = new PointLight(make_float3(4.0f, 10.0f, -1.5f), 1.3f);
	w->scene_lights[1] = lt2;

	DirectionalLight* lt3 = new DirectionalLight(make_float3(-1.0f, -1.0f, -2.0f), 1.0f);
	w->scene_lights[2] = lt3;

	Matte* mt1 = new Matte(w->pixel_sampler, 0.3f, 0.7f, make_float3(0.6f, 0.6f, 0.6f));
	w->scene_materials[0] = mt1;
	Plane * ptr1 = new Plane();
	ptr1->init(make_float3(0, -8.0f, 0), make_float3(0, 1, 0), mt1);
	w->scene_objs[0] = ptr1;

	Phong* mt2 = new Phong(w->pixel_sampler, 0.3f, 0.8f, 0.1f, 15.0f, make_float3(0.9f, 0.001f, 0.001f));
	w->scene_materials[1] = mt2;
	Sphere * ptr2 = new Sphere();
	ptr2->init(make_float3(0, 0, 0), 4, mt2);
	w->scene_objs[1] = ptr2;
}

void buildScene(World *w) {
	printf("Building scene\n"); fflush(stdout);

	w->num_of_objects = 2;
	w->num_of_lights = 3;
	w->num_of_materials = w->num_of_objects;

	hipMalloc(&w->scene_objs, sizeof(GObject*) * w->num_of_objects);
	hipMalloc(&w->scene_lights, sizeof(Light*) * w->num_of_lights);
	hipMalloc(&w->scene_materials, sizeof(Material*) * w->num_of_materials);

	buildScene_gpu <<< 1, 1 >>> (w);
	SYNC_AND_CHECK_CUDA_ERRORS;

	printf("\tOK\n"); fflush(stdout);
}
