#include "hip/hip_runtime.h"
#include "World.h"
#include <cstdio>
#include "hiprand.h"
#include "hiprand/hiprand_kernel.h"

__global__ void Render_Frame(World * w, int offset) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int b_idx = threadIdx.x;
	hiprandState_t state;
	hiprand_init(23041996 * idx, 0, 0, &state);

	__shared__ int width; if (b_idx == 0) width = w->vp.res.x;
	__shared__ int height; if (b_idx == 0) height = w->vp.res.y;
	__shared__ float inv_sampler_points_num; if (b_idx == 0) inv_sampler_points_num = 1.0f / w->num_rays_per_pixel;
	__syncthreads();
	if (idx + offset < width * height) {
		int x = (idx + offset) % width;
		int y = (idx + offset) / width;

		Ray ray;
		HitRec hr;
		float2 pt;
		float3 color;

		int sampler_set_idx = hiprand(&state) % w->pixel_sampler->num_sets;

		w->gpu_buffer[idx] = make_float3(0, 0, 0);

		for (int i = 0; i < w->num_rays_per_pixel; ++i){
			ray.sampler_point_number = i;
			ray.sampler_set_number = sampler_set_idx;

			pt = w->pixel_sampler->SampleUnitSquare(ray.sampler_point_number, ray.sampler_set_number);

			w->camera->MakeRay(&ray, make_int2(x, y), pt);

			make_HitRec(&hr, w, &ray);

			w->ray_tracer->TraceRay(&ray, &hr);

			color = w->background_color;

			if (hr.isHit)
				color = hr.material_ptr->shade(&hr);

			w->gpu_buffer[idx] += color * inv_sampler_points_num;
			__syncthreads();
		}
	}
}

void renderScene(World *w, int pixels_in_frame) {
	printf("Rendering scene:\n\t");
	printf("Resolution: %d x %d\n\t", w->vp.res.x, w->vp.res.y);
	printf("Sampling: %d\n\t", w->num_rays_per_pixel);
	printf("Number of objects: %d\n\t", w->num_of_objects);
	printf("Number of lights: %d\n\t", w->num_of_lights);
	printf("Frame size: %d\n\t", pixels_in_frame);
	fflush(stdout);

	hipMalloc(&w->gpu_buffer, sizeof(float3) * pixels_in_frame);

	int block_size = BLOCKSIZE;
	int num_pixels = w->vp.res.x * w->vp.res.y;
	int num_frames = num_pixels / pixels_in_frame + (num_pixels % pixels_in_frame ? 1 : 0);

	printf("Frame number: %d\n", num_frames); fflush(stdout);

	int pixels_rendered = 0;
	for (int frame_idx = 0; frame_idx < num_frames; ++frame_idx) {
		printf("\r\tTotal: %.2f %%", (float)(pixels_rendered) / num_pixels * 100.0f);fflush(stdout);
		int pixels_to_render = (pixels_in_frame < num_pixels - pixels_rendered ? pixels_in_frame : num_pixels - pixels_rendered);
		int num_blocks = pixels_to_render / block_size + (pixels_to_render % block_size ? 1 : 0);

		Render_Frame <<< num_blocks, block_size >>> (w, pixels_rendered);
		SYNC_AND_CHECK_CUDA_ERRORS;

		hipMemcpy(w->image + pixels_rendered, w->gpu_buffer, sizeof(float3) * pixels_to_render, hipMemcpyDeviceToHost);
		SYNC_AND_CHECK_CUDA_ERRORS;

		pixels_rendered += pixels_to_render;
	}
	printf("\r");fflush(stdout);

	hipFree(w->gpu_buffer);
}
