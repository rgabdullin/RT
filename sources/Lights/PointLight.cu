#include "hip/hip_runtime.h"
#include "PointLight.h"
#include "../Utils/cuda_math.h"

/* PointLight */
__device__
PointLight::PointLight(float3 l_location, float l_ls, float3 l_color) : ls(l_ls), color(l_color), location(l_location) {}
__device__
float3 PointLight::GetDirection(HitRec* hr) {
	return normalize(location - hr->hit_point);
}
__device__
float3 PointLight::L(HitRec* hr) {
	return color * ls;
}
__device__
PointLight::~PointLight() {}
