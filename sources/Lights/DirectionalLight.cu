#include "hip/hip_runtime.h"
#include "DirectionalLight.h"
#include "../Utils/cuda_math.h"

/* DirectionLight */
__device__
DirectionalLight::DirectionalLight(float3 l_direction, float l_ls, float3 l_color) : ls(l_ls), color(l_color), direction(-1.0f * normalize(l_direction)) {}
__device__
float3 DirectionalLight::GetDirection(HitRec* hr) {
	return direction;
}
__device__
float3 DirectionalLight::L(HitRec* hr) {
	return color * ls;
}
__device__
DirectionalLight::~DirectionalLight() {}
