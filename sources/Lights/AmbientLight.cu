#include "hip/hip_runtime.h"
#include "AmbientLight.h"
#include "../Utils/cuda_math.h"

/* AmbientLight */
__device__
AmbientLight::AmbientLight(float l_ls, float3 l_color) : ls(l_ls), color(l_color) {}
__device__
float3 AmbientLight::GetDirection(HitRec* hr) {
	return make_float3(0.0f, 0.0f, 0.0f);
}
__device__
float3 AmbientLight::L(HitRec* hr) {
	return color * ls;
}
__device__
AmbientLight::~AmbientLight() {}
