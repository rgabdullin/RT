#include "hip/hip_runtime.h"
#include "PinholeCamera.h"
#include "../Utils/World.h"
#include "../Utils/cuda_math.h"

/* Pinhole Camera */

__device__
PinholeCamera::PinholeCamera(World* l_wr) : Camera(l_wr), dist_to_vp(1.0), zoom(1.0) {}

__device__
void PinholeCamera::init_pinhole(float l_dist_to_vp, float l_zoom) {
	dist_to_vp = l_dist_to_vp;
	zoom = l_zoom;
}

__device__
void PinholeCamera::MakeRay(Ray* ray, int2 pixel, float2 sample) {
	ray->origin = eye;
	float2 pixel_point = make_float2((float(pixel.x) + sample.x) * wr->vp.psize - 0.5 * wr->vp.psize * wr->vp.res.x, (float(pixel.y) + sample.y)* wr->vp.psize - 0.5 * wr->vp.psize * wr->vp.res.y);
	ray->direction = normalize(-dist_to_vp * zoom * w + pixel_point.x * u + pixel_point.y * v);
	ray->image_idx = pixel.x + pixel.y * wr->vp.res.x;
}
__device__
PinholeCamera::~PinholeCamera() {}
