#include "hip/hip_runtime.h"
/* Orthographic Camera */
#include "OrthographicCamera.h"
#include "../Utils/World.h"
#include "../Utils/cuda_math.h"

__device__
OrthographicCamera::OrthographicCamera(World* l_wr) : Camera(l_wr) {}

__device__
void OrthographicCamera::MakeRay(Ray* ray, int2 pixel, float2 sample) {
	ray->origin = eye
		+ ((float(pixel.x) + sample.x) * wr->vp.psize - 0.5 * wr->vp.psize * wr->vp.res.x) * u
		+ ((float(pixel.y) + sample.y) * wr->vp.psize - 0.5 * wr->vp.psize * wr->vp.res.y) * v;
	ray->direction = normalize(-w);
	ray->image_idx = pixel.x + pixel.y * wr->vp.res.x;
}
__device__
OrthographicCamera::~OrthographicCamera() {}
