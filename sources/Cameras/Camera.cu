#include "hip/hip_runtime.h"
#include "Camera.h"
#include "../Utils/Ray.h"
#include "../Utils/World.h"
#include "../Utils/cuda_math.h"

/* Camera */

__device__
Camera::Camera(World* l_wr) {
	wr = l_wr;
}

__device__
void Camera::init(float3 l_eye, float3 l_lookat, float3 l_up) {
	eye = l_eye;
	lookat = l_lookat;
	up = l_up;
	w = normalize(eye - lookat);
	u = normalize(cross(up, w));
	v = cross(w, u);
}

__device__
Camera::~Camera() {}
